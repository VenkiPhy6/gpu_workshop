// matrix_multiplication_cuda.cu
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <chrono>

void checkCuda(hipError_t result, const char *msg) {
    if (result != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(result) << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char* argv[]) {
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " N\n";
        return 1;
    }

    int N = std::atoi(argv[1]);
    size_t size = static_cast<size_t>(N) * N;

    // Allocate host memory
    float *h_A = new float[size];
    float *h_B = new float[size];
    float *h_C = new float[size];

    // Initialize matrices with random values
    for (size_t i = 0; i < size; ++i) {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    checkCuda(hipMalloc(&d_A, size * sizeof(float)), "hipMalloc A");
    checkCuda(hipMalloc(&d_B, size * sizeof(float)), "hipMalloc B");
    checkCuda(hipMalloc(&d_C, size * sizeof(float)), "hipMalloc C");

    // Copy host to device
    checkCuda(hipMemcpy(d_A, h_A, size * sizeof(float), hipMemcpyHostToDevice), "memcpy A");
    checkCuda(hipMemcpy(d_B, h_B, size * sizeof(float), hipMemcpyHostToDevice), "memcpy B");

    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float alpha = 1.0f, beta = 0.0f;

    auto start = std::chrono::high_resolution_clock::now();

    // Matrix multiplication: C = alpha * A * B + beta * C
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                N, N, N,
                &alpha,
                d_A, N,
                d_B, N,
                &beta,
                d_C, N);

    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> elapsed = end - start;
    std::cout << "N=" << N << ", time=" << elapsed.count() << " seconds" << std::endl;

    // Cleanup
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}
